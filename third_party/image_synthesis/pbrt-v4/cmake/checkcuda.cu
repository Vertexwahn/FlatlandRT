// https://wagonhelm.github.io/articles/2018-03/detecting-cuda-capability-with-cmake
// Justin Francis


#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char **argv){
    hipDeviceProp_t dP;
    float min_cc = 5.2;

    int rc = hipGetDeviceProperties(&dP, 0);
    if(rc != hipSuccess) {
        hipError_t error = hipGetLastError();
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return rc; /* Failure */
    }
    float cc = dP.major + (dP.minor / 10.0);
    if(cc < min_cc) {
        printf("Minimum Compute Capability of %2.1f required: %2.1f found. Not Building CUDA Code.\n",
               min_cc, cc);
        return 1; /* Failure */
    } else {
        printf("sm_%d%d", dP.major, dP.minor);
        return 0; /* Success */
    }
}
